
#include <hip/hip_runtime.h>
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef PADDLE_WITH_HETERPS
#include <algorithm>
#include <ctime>
#include <memory>
#include <numeric>
#include "paddle/fluid/framework/fleet/heter_ps/optimizer_conf.h"
#include "paddle/fluid/framework/lod_tensor.h"
#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/fluid/framework/fleet/heter_ps/feature_value.h"
namespace paddle {
namespace framework {

template <typename GPUAccessor>
__global__ void PullCopy(float** dest,
                         const float* src,
                         const int64_t* len,
                         int slot_num,
                         int total_len,
                         uint64_t** keys,
                         uint64_t max_val_size,
                         int* gpu_dim,
                         GPUAccessor gpu_accessor) {
  
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);

    float* feature_value_ptr =
        (float*)((char*)src + uint64_t(i) * uint64_t(max_val_size));
    int mf_dim = gpu_dim[x] - 3;

    gpu_accessor.Select(
        dest[x] + y * (mf_dim + 3), feature_value_ptr, keys[x] + y, mf_dim);
  }
}

template <typename GPUAccessor>
__global__ void PushCopyWithPool(float* dest,
                                 float** src,
                                 int64_t* len,
                                 int slot_num,
                                 uint64_t total_len,
                                 int bs,
                                 int* slot_vector,
                                 int* mf_dim_vector,
                                 size_t grad_value_size,
                                 GPUAccessor gpu_accessor) {

  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[low - 1] : 0);
    float* cur =
        (float*)((char*)dest + i * grad_value_size);
   
    int mf_dim = mf_dim_vector[x];  // slot_vector holds both slot and
                                    // slot:mf_dim information
    gpu_accessor.GradientSelect(cur, src[x] + y * (mf_dim + 3), slot_vector[x], mf_dim, bs);
  }
}

template <typename GPUAccessor>
void AccessorWrapper<GPUAccessor>::CopyForPull(const paddle::platform::Place& place,
                                               uint64_t** gpu_keys,
                                               const std::vector<float*>& values,
                                               const float* total_values_gpu,
                                               const int64_t* gpu_len,
                                               const int slot_num,
                                               const int hidden_size,
                                               const int64_t total_length,
                                               int* gpu_dim,
                                               size_t val_type_size) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(place))
                    ->stream();
  auto buf_value = memory::Alloc(place, values.size() * sizeof(float*));
  float** gpu_values = reinterpret_cast<float**>(buf_value->ptr());
  cudaMemcpy(gpu_values, values.data(), values.size() * sizeof(float*),
             cudaMemcpyHostToDevice);
  PullCopy<<<(total_length + 1024 - 1) / 1024, 1024, 0, stream>>>(
        gpu_values, total_values_gpu, gpu_len, slot_num, total_length, gpu_keys,
        val_type_size, gpu_dim, gpu_accessor_);
  cudaStreamSynchronize(stream);
}

template <typename GPUAccessor>
void AccessorWrapper<GPUAccessor>::CopyForPush(const paddle::platform::Place& place,
                                               const std::vector<const float*>& grad_values,
                                               float* total_grad_values_gpu,
                                               const std::vector<int64_t>& slot_lengths,
                                               const uint64_t total_length,
                                               const int batch_size,
                                               size_t grad_value_size,
                                               std::vector<int>& slot_vector,
                                               std::vector<int>& slot_mf_dim_vector) {

  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(place))
                    ->stream();
  auto slot_lengths_lod = slot_lengths;
  for (int i = 1; i < slot_lengths_lod.size(); i++) {
    slot_lengths_lod[i] += slot_lengths_lod[i - 1];
  }
  auto buf_grad_value =
      memory::Alloc(place, grad_values.size() * sizeof(float*));
  auto buf_length =
      memory::Alloc(place, slot_lengths.size() * sizeof(int64_t));
  auto buf_slot_vector =
      memory::Alloc(place, slot_lengths_lod.size() * sizeof(int));
  auto buf_mf_dim_vector =
      memory::Alloc(place, slot_lengths_lod.size() * sizeof(int));

  float** gpu_values = reinterpret_cast<float**>(buf_grad_value->ptr());
  int64_t* gpu_len = reinterpret_cast<int64_t*>(buf_length->ptr());
  int* d_slot_vector = reinterpret_cast<int*>(buf_slot_vector->ptr());
  int* d_mf_dim_vector = reinterpret_cast<int*>(buf_mf_dim_vector->ptr());

  cudaMemcpy(gpu_values, grad_values.data(),
             grad_values.size() * sizeof(float*), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_len, slot_lengths_lod.data(),
             slot_lengths.size() * sizeof(int64_t), cudaMemcpyHostToDevice);
  cudaMemcpy(d_slot_vector, slot_vector.data(),
             slot_lengths_lod.size() * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(d_mf_dim_vector, slot_mf_dim_vector.data(),
             slot_lengths_lod.size() * sizeof(int), cudaMemcpyHostToDevice);

  PushCopyWithPool<<<(total_length + 1024 - 1) / 1024, 1024, 0, stream>>>(
      total_grad_values_gpu, gpu_values, gpu_len, slot_lengths.size(),
      total_length, batch_size, d_slot_vector, d_mf_dim_vector,
      grad_value_size, gpu_accessor_);

  cudaStreamSynchronize(stream);
}

#ifdef PADDLE_WITH_PSLIB
template class AccessorWrapper<CommonFeatureValueAccessor>;
#endif



}  // end namespace framework
}  // end namespace paddle
#endif